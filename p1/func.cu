#include "hip/hip_runtime.h"
//****************************************************************************
// Also note that we've supplied a helpful debugging function called checkCudaErrors.
// You should wrap your allocation and copying statements like we've done in the
// code we're supplying you. Here is an example of the unsafe way to allocate
// memory on the GPU:
//
// hipMalloc(&d_red, sizeof(unsigned char) * numRows * numCols);
//
// Here is an example of the safe way to do the same thing:
//
// checkCudaErrors(hipMalloc(&d_red, sizeof(unsigned char) * numRows * numCols));
//****************************************************************************

#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""

#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
	//system("pause");
    exit(1);
  }
}

#define TAMFILTRO 5
#define R 2
#define BLOCKSIZE 32
#define SHAREDBLOCKSIZE 36


//Usamos como memoria constante el filtro de entrada
__constant__ float d_const_filter[TAMFILTRO*TAMFILTRO];

__global__
void shared_box_filter(const unsigned char* const inputChannel,
unsigned char* const outputChannel,
int numRows, int numCols, const int filterWidth)
{
	__shared__ float sharedMem[SHAREDBLOCKSIZE][SHAREDBLOCKSIZE];
	
	const int2 thread_2D_pos = make_int2(blockIdx.x * (blockDim.x) + threadIdx.x,
		blockIdx.y * (blockDim.y) + threadIdx.y);

	const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

	int sharedPosX = (threadIdx.x);
	int sharedPosY = (threadIdx.y);

	if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
		return;

	if (sharedPosY == 0){
		sharedMem[sharedPosY][sharedPosX + 2] = inputChannel[thread_1D_pos];
		sharedMem[sharedPosY + 1][sharedPosX + 2] = inputChannel[thread_1D_pos];
	}
	else if (sharedPosX == 0){
		sharedMem[sharedPosY + 2][sharedPosX] = inputChannel[thread_1D_pos];
		sharedMem[sharedPosY + 2][sharedPosX + 1] = inputChannel[thread_1D_pos];
	}
	else if (sharedPosY == BLOCKSIZE - 1){
		sharedMem[sharedPosY + 3][sharedPosX + 2] = inputChannel[thread_1D_pos];
		sharedMem[sharedPosY + 4][sharedPosX + 2] = inputChannel[thread_1D_pos];
	}
	else if (sharedPosX == BLOCKSIZE - 1){
		sharedMem[sharedPosY + 2][sharedPosX + 3] = inputChannel[thread_1D_pos];
		sharedMem[sharedPosY + 2][sharedPosX + 4] = inputChannel[thread_1D_pos];
	}

	if (sharedPosX == 0 && sharedPosY == 0){
		sharedMem[sharedPosY][sharedPosX] = inputChannel[thread_1D_pos];
		sharedMem[sharedPosY][sharedPosX + 1] = inputChannel[thread_1D_pos];
		sharedMem[sharedPosY + 1][sharedPosX] = inputChannel[thread_1D_pos];
		sharedMem[sharedPosY + 1][sharedPosX + 1] = inputChannel[thread_1D_pos];
		
	}
	else if (sharedPosX == BLOCKSIZE - 1 && sharedPosY == 0){
		sharedMem[sharedPosY][sharedPosX + 4] = inputChannel[thread_1D_pos];
		sharedMem[sharedPosY][sharedPosX + 3] = inputChannel[thread_1D_pos];
		sharedMem[sharedPosY + 1][sharedPosX + 4] = inputChannel[thread_1D_pos];
		sharedMem[sharedPosY + 1][sharedPosX + 3] = inputChannel[thread_1D_pos];
	}
	else if (sharedPosX == 0 && sharedPosY == BLOCKSIZE - 1){
		sharedMem[sharedPosY + 4][sharedPosX] = inputChannel[thread_1D_pos];
		sharedMem[sharedPosY + 3][sharedPosX] = inputChannel[thread_1D_pos];
		sharedMem[sharedPosY + 4][sharedPosX + 1] = inputChannel[thread_1D_pos];
		sharedMem[sharedPosY + 3][sharedPosX + 1] = inputChannel[thread_1D_pos];
	}
	else if (sharedPosX == BLOCKSIZE - 1 && sharedPosY == BLOCKSIZE - 1){
		sharedMem[sharedPosY + 4][sharedPosX + 4] = inputChannel[thread_1D_pos];
		sharedMem[sharedPosY + 3][sharedPosX + 4] = inputChannel[thread_1D_pos];
		sharedMem[sharedPosY + 4][sharedPosX + 3] = inputChannel[thread_1D_pos];
		sharedMem[sharedPosY + 3][sharedPosX + 3] = inputChannel[thread_1D_pos];
	}
		
	sharedMem[sharedPosY + R][sharedPosX + R] = inputChannel[thread_1D_pos];

	__syncthreads();

	int posFiltro = 0;
	float result = 0.0f;

	//Recorremos las posiciones del filtro
	for (int filter_r = 0; filter_r <= R * 2; ++filter_r){
		for (int filter_c = 0; filter_c <= R * 2; ++filter_c){
			float image_value = sharedMem[sharedPosY + filter_r][sharedPosX + filter_c];
			float filter_value = d_const_filter[posFiltro];
			result += image_value * filter_value;
			posFiltro++;
		}
	}

	outputChannel[thread_1D_pos] = result;
}

__global__
void box_filter(const unsigned char* const inputChannel,
                   unsigned char* const outputChannel,
                   int numRows, int numCols, const int filterWidth)
{
	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);

	const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

	if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
		return;

	int posFil = 0;
	float result = 0.0f;

	//Recorremos las posiciones del filtro
	for (int filter_r = -filterWidth / 2; filter_r <= filterWidth / 2; ++filter_r){
		for (int filter_c = -filterWidth / 2; filter_c <= filterWidth / 2; ++filter_c){

			int image_r = thread_2D_pos.y + filter_r;
			int image_c = thread_2D_pos.x + filter_c;

			//Comprobamos que la posicion del filtro se encuentra dentro de las posiciones de la imagen
			if ((image_c >= 0) && (image_c < numCols) && (image_r >= 0) && (image_r < numRows)){
				float image_value = inputChannel[image_r * numCols + image_c];
				float filter_value = d_const_filter[posFil];
				result += image_value * filter_value;
			}
			posFil++;
		}
	}

	outputChannel[thread_1D_pos] = result;
}

//This kernel takes in an image represented as a uchar4 and splits
//it into three images consisting of only one color channel each
__global__
void separateChannels(const uchar4* const inputImageRGBA,
                      int numRows,
                      int numCols,
                      unsigned char* const redChannel,
                      unsigned char* const greenChannel,
                      unsigned char* const blueChannel)
{
	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);

	const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

	if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
		return;

	//Separamos en tres canales la imagen
	int id = thread_1D_pos;
	redChannel[id] = inputImageRGBA[id].x;
	greenChannel[id] = inputImageRGBA[id].y;
	blueChannel[id] = inputImageRGBA[id].z;
}

//This kernel takes in three color channels and recombines them
//into one image. The alpha channel is set to 255 to represent
//that this image has no transparency.
__global__
void recombineChannels(const unsigned char* const redChannel,
                       const unsigned char* const greenChannel,
                       const unsigned char* const blueChannel,
                       uchar4* const outputImageRGBA,
                       int numRows,
                       int numCols)
{
  const int2 thread_2D_pos = make_int2( blockIdx.x * blockDim.x + threadIdx.x,
                                        blockIdx.y * blockDim.y + threadIdx.y);

  const int thread_1D_pos = thread_2D_pos.y * numCols + thread_2D_pos.x;

  //make sure we don't try and access memory outside the image
  //by having any threads mapped there return early
  if (thread_2D_pos.x >= numCols || thread_2D_pos.y >= numRows)
    return;

  unsigned char red   = redChannel[thread_1D_pos];
  unsigned char green = greenChannel[thread_1D_pos];
  unsigned char blue  = blueChannel[thread_1D_pos];

  //Alpha should be 255 for no transparency
  uchar4 outputPixel = make_uchar4(red, green, blue, 255);

  outputImageRGBA[thread_1D_pos] = outputPixel;
}

unsigned char *d_red, *d_green, *d_blue;
float         *d_filter;

void allocateMemoryAndCopyToGPU(const size_t numRowsImage, const size_t numColsImage,
                                const float* const h_filter, const size_t filterWidth)
{

  //allocate memory for the three different channels
  checkCudaErrors(hipMalloc(&d_red,   sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_green, sizeof(unsigned char) * numRowsImage * numColsImage));
  checkCudaErrors(hipMalloc(&d_blue,  sizeof(unsigned char) * numRowsImage * numColsImage));

  //TODO:
  //Reservar memoria para el filtro en GPU: d_filter, la cual ya esta declarada
  // Copiar el filtro  (h_filter) a memoria global de la GPU (d_filter)

 // checkCudaErrors(hipMalloc(&d_filter, sizeof(unsigned char) * filterWidth * filterWidth));
  //checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_const_filter), h_filter, sizeof(unsigned char) * filterWidth * filterWidth));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_const_filter), h_filter, sizeof(float) * filterWidth * filterWidth));
 // hipMemcpy(d_filter, h_filter, sizeof(unsigned char) * filterWidth * filterWidth, hipMemcpyHostToDevice);//Copiamos el d_filter a GPU.

}


void create_filter(float **h_filter, int *filterWidth){

  const int KernelWidth = 5; //OJO CON EL TAMA�O DEL FILTRO//
  *filterWidth = KernelWidth;

  //create and fill the filter we will convolve with
  *h_filter = new float[KernelWidth * KernelWidth];
  
  
  //Filtro gaussiano: blur
  //const float KernelSigma = 2.;

  //float filterSum = 0.f; //for normalization

  //for (int r = -KernelWidth/2; r <= KernelWidth/2; ++r) {
  //  for (int c = -KernelWidth/2; c <= KernelWidth/2; ++c) {
  //    float filterValue = expf( -(float)(c * c + r * r) / (2.f * KernelSigma * KernelSigma));
  //    (*h_filter)[(r + KernelWidth/2) * KernelWidth + c + KernelWidth/2] = filterValue;
  //    filterSum += filterValue;
  //  }
  //}

  //float normalizationFactor = 1.f / filterSum;

  //for (int r = -KernelWidth/2; r <= KernelWidth/2; ++r) {
  //  for (int c = -KernelWidth/2; c <= KernelWidth/2; ++c) {
  //    (*h_filter)[(r + KernelWidth/2) * KernelWidth + c + KernelWidth/2] *= normalizationFactor;
  //  }
  //}
  

  ////Laplaciano 5x5
  (*h_filter)[0] = 0;   (*h_filter)[1] = 0;    (*h_filter)[2] = -1.;  (*h_filter)[3] = 0;    (*h_filter)[4] = 0;
  (*h_filter)[5] = 1.;  (*h_filter)[6] = -1.;  (*h_filter)[7] = -2.;  (*h_filter)[8] = -1.;  (*h_filter)[9] = 0;
  (*h_filter)[10] = -1.;(*h_filter)[11] = -2.; (*h_filter)[12] = 17.; (*h_filter)[13] = -2.; (*h_filter)[14] = -1.;
  (*h_filter)[15] = 1.; (*h_filter)[16] = -1.; (*h_filter)[17] = -2.; (*h_filter)[18] = -1.; (*h_filter)[19] = 0;
  (*h_filter)[20] = 0; (*h_filter)[21] = 0;   (*h_filter)[22] = -1.; (*h_filter)[23] = 0;   (*h_filter)[24] = 0;
  
  //ESTOS DOS M�TODOS LOS HE SACADO DE INTERNET Y DAN UN RESULTADO PARECIDO. Se aprecia la eliminaci�n de ruido al hacer zoom! Comprobar con la original

  ////Filtro de baja frecuencia (paso bajo) = desenfoque, interpolaci�n, eliminaci�n de ruido
  //(*h_filter)[0] = 1. / 25; (*h_filter)[1] = 1. / 25; (*h_filter)[2] = 1. / 25; (*h_filter)[3] = 1. / 25; (*h_filter)[4] = 1. / 25;
  //(*h_filter)[5] = 1. / 25; (*h_filter)[6] = 1. / 25; (*h_filter)[7] = 1. / 25; (*h_filter)[8] = 1. / 25; (*h_filter)[9] = 1. / 25;
  //(*h_filter)[10] = 1. / 25; (*h_filter)[11] = 1. / 25; (*h_filter)[12] = 1. / 25; (*h_filter)[13] = 1. / 25; (*h_filter)[14] = 1. / 25;
  //(*h_filter)[15] = 1. / 25; (*h_filter)[16] = 1. / 25; (*h_filter)[17] = 1. / 25; (*h_filter)[18] = 1. / 25; (*h_filter)[19] = 1. / 25;
  //(*h_filter)[20] = 1. / 25; (*h_filter)[21] = 1. / 25; (*h_filter)[22] = 1. / 25; (*h_filter)[23] = 1. / 25; (*h_filter)[24] = 1. / 25;
  
  //Interpolaci�n ponderada
  /*(*h_filter)[0] = 1. / 36; (*h_filter)[1] = 1. / 36; (*h_filter)[2] = 1. / 36; (*h_filter)[3] = 1. / 36; (*h_filter)[4] = 1. / 36;
  (*h_filter)[5] = 1. / 36; (*h_filter)[6] = 2. / 36; (*h_filter)[7] = 2. / 36; (*h_filter)[8] = 2. / 36;  (*h_filter)[9] = 1. / 36;
  (*h_filter)[10] = 1. / 36; (*h_filter)[11] = 2. / 36; (*h_filter)[12] = 4. / 36; (*h_filter)[13] = 2. / 36; (*h_filter)[14] = 1. / 36;
  (*h_filter)[15] = 1. / 36; (*h_filter)[16] = 2. / 36; (*h_filter)[17] = 2. / 36; (*h_filter)[18] = 2. / 36; (*h_filter)[19] = 1. / 36;
  (*h_filter)[20] = 1. / 36; (*h_filter)[21] = 1. / 36; (*h_filter)[22] = 1. / 36; (*h_filter)[23] = 1. / 36; (*h_filter)[24] = 1. / 36;*/

  //Filtro paso bajo = suavizado
  /*(*h_filter)[0] = 1.; (*h_filter)[1] = 1.; (*h_filter)[2] = 1.; (*h_filter)[3] = 1.; (*h_filter)[4] = 1.;
  (*h_filter)[5] = 1.; (*h_filter)[6] = 4.; (*h_filter)[7] = 4.; (*h_filter)[8] = 4.;  (*h_filter)[9] = 1.;
  (*h_filter)[10] = 1.; (*h_filter)[11] = 4.; (*h_filter)[12] = 12.; (*h_filter)[13] = 4.; (*h_filter)[14] = 1.;
  (*h_filter)[15] = 1.; (*h_filter)[16] = 4.; (*h_filter)[17] = 4.; (*h_filter)[18] = 4.; (*h_filter)[19] = 1.;
  (*h_filter)[20] = 1.; (*h_filter)[21] = 1.; (*h_filter)[22] = 1.; (*h_filter)[23] = 1.; (*h_filter)[24] = 1.;*/

  //Filtro Gaussiano
  /*(*h_filter)[0] = 2.; (*h_filter)[1] = 4.; (*h_filter)[2] = 7.; (*h_filter)[3] = 4.; (*h_filter)[4] = 1.;
  (*h_filter)[5] = 4.; (*h_filter)[6] = 9.; (*h_filter)[7] = 12.; (*h_filter)[8] = 9.; (*h_filter)[9] = 4.;
  (*h_filter)[10] = 5.; (*h_filter)[11] = 12.; (*h_filter)[12] = 15.; (*h_filter)[13] = 12.; (*h_filter)[14] = 5.;
  (*h_filter)[15] = 4.; (*h_filter)[16] = 9.; (*h_filter)[17] = 12.; (*h_filter)[18] = 9.; (*h_filter)[19] = 4.;
  (*h_filter)[20] = 2.; (*h_filter)[21] = 4.; (*h_filter)[22] = 5.; (*h_filter)[23] = 4.; (*h_filter)[24] = 1.;
*/
  //Filtro de nitidez 
  /*(*h_filter)[0] = -1.; (*h_filter)[1] = -3.; (*h_filter)[2] = -4.; (*h_filter)[3] = -3.; (*h_filter)[4] = -1.;
  (*h_filter)[5] = -3.; (*h_filter)[6] = 0.; (*h_filter)[7] = 6.; (*h_filter)[8] = 0.;  (*h_filter)[9] = -3.;
  (*h_filter)[10] = -4.; (*h_filter)[11] = 6.; (*h_filter)[12] = 21.; (*h_filter)[13] = 6.; (*h_filter)[14] = -4.;
  (*h_filter)[15] = -3.; (*h_filter)[16] = 0.; (*h_filter)[17] = 6.; (*h_filter)[18] = 0.; (*h_filter)[19] = -3.;
  (*h_filter)[20] = -1.; (*h_filter)[21] = -3.; (*h_filter)[22] = -4.; (*h_filter)[23] = -3.; (*h_filter)[24] = -1.;*/

  //Detecci�n de bordes
 /* (*h_filter)[0] = 0.; (*h_filter)[1] = 0.; (*h_filter)[2] = 0.; (*h_filter)[3] = 0.; (*h_filter)[4] = 0.;
  (*h_filter)[5] = 0.; (*h_filter)[6] = 0.; (*h_filter)[7] = 1.; (*h_filter)[8] = 0.;  (*h_filter)[9] = 0.;
  (*h_filter)[10] = 0.; (*h_filter)[11] = 1.; (*h_filter)[12] = -4.; (*h_filter)[13] = 1.; (*h_filter)[14] = 0.;
  (*h_filter)[15] = 0.; (*h_filter)[16] = 0.; (*h_filter)[17] = 1.; (*h_filter)[18] = 0.; (*h_filter)[19] = 0.;
  (*h_filter)[20] = 0.; (*h_filter)[21] = 0.; (*h_filter)[22] = 0.; (*h_filter)[23] = 0.; (*h_filter)[24] = 0.;*/

  /*(*h_filter)[0] = 0.; (*h_filter)[1] = 0.; (*h_filter)[2] = 0.; (*h_filter)[3] = 0.; (*h_filter)[4] = 0.;
  (*h_filter)[5] = 0.; (*h_filter)[6] = -1.; (*h_filter)[7] = -1.; (*h_filter)[8] = -1.;  (*h_filter)[9] = 0.;
  (*h_filter)[10] = 0.; (*h_filter)[11] = -1.; (*h_filter)[12] = 8.; (*h_filter)[13] = -1.; (*h_filter)[14] = 0.;
  (*h_filter)[15] = 0.; (*h_filter)[16] = -1.; (*h_filter)[17] = -1.; (*h_filter)[18] = -1.; (*h_filter)[19] = 0.;
  (*h_filter)[20] = 0.; (*h_filter)[21] = 0.; (*h_filter)[22] = 0.; (*h_filter)[23] = 0.; (*h_filter)[24] = 0.;*/

  //Es bastante suave
  /*(*h_filter)[0] = 0.; (*h_filter)[1] = 0.; (*h_filter)[2] = 0.; (*h_filter)[3] = 0.; (*h_filter)[4] = 0.;
  (*h_filter)[5] = 0.; (*h_filter)[6] = -1.; (*h_filter)[7] = 0.; (*h_filter)[8] = 1.;  (*h_filter)[9] = 0.;
  (*h_filter)[10] = 0.; (*h_filter)[11] = -1.; (*h_filter)[12] = 0.; (*h_filter)[13] = 1.; (*h_filter)[14] = 0.;
  (*h_filter)[15] = 0.; (*h_filter)[16] = -1.; (*h_filter)[17] = 0.; (*h_filter)[18] = 1.; (*h_filter)[19] = 0.;
  (*h_filter)[20] = 0.; (*h_filter)[21] = 0.; (*h_filter)[22] = 0.; (*h_filter)[23] = 0.; (*h_filter)[24] = 0.;*/
  //TODO: crear los filtros segun necesidad
  //NOTA: cuidado al establecer el tama�o del filtro a utilizar

}


void convolution(const uchar4 * const h_inputImageRGBA, uchar4 * const d_inputImageRGBA,
                        uchar4* const d_outputImageRGBA, const size_t numRows, const size_t numCols,
                        unsigned char *d_redFiltered, 
                        unsigned char *d_greenFiltered, 
                        unsigned char *d_blueFiltered,
                        const int filterWidth)
{
  //TODO: Calcular tama�os de bloque
  const dim3 blockSize(BLOCKSIZE, BLOCKSIZE, 1);
  //const dim3 gridSize((numCols / (blockSize.x - 2 * R)) + 1, (numRows / (blockSize.y - 2 * R)) + 1, 1);
  const dim3 gridSize1((numCols / (blockSize.x)) + 1, (numRows / (blockSize.y)) + 1, 1);

  //TODO: Lanzar kernel para separar imagenes RGBA en diferentes colores
  separateChannels << < gridSize1, blockSize >> >(d_inputImageRGBA, numRows, numCols, d_redFiltered, d_greenFiltered, d_blueFiltered);

  //TODO: Ejecutar convoluci�n. Una por canal
  //box_filter << <gridSize1, blockSize >> > (d_redFiltered, d_red, numRows, numCols, filterWidth);
  //box_filter << <gridSize1, blockSize >> > (d_greenFiltered, d_green, numRows, numCols, filterWidth);
  //box_filter << <gridSize1, blockSize >> > (d_blueFiltered, d_blue, numRows, numCols, filterWidth);
  ////Memoria compartida
  shared_box_filter << <gridSize1, blockSize >> > (d_redFiltered, d_red, numRows, numCols, filterWidth);
  shared_box_filter << <gridSize1, blockSize >> > (d_greenFiltered, d_green, numRows, numCols, filterWidth);
  shared_box_filter << <gridSize1, blockSize >> > (d_blueFiltered, d_blue, numRows, numCols, filterWidth);

  // Recombining the results. 
  //recombineChannels << <gridSize, blockSize >> >(d_redFiltered, d_greenFiltered, d_blueFiltered, d_outputImageRGBA, numRows, numCols);
  recombineChannels << <gridSize1, blockSize >> >(d_red, d_green, d_blue, d_outputImageRGBA, numRows, numCols);

  hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());
  //system("pause");
}


//Free all the memory that we allocated
//TODO: make sure you free any arrays that you allocated
void cleanup() {
  checkCudaErrors(hipFree(d_red));
  checkCudaErrors(hipFree(d_green));
  checkCudaErrors(hipFree(d_blue));
}
